#include "hip/hip_runtime.h"
#include "mc/mc.cuh"
#include "utils.cuh"

#include <nanobind/nanobind.h>
#include <nanobind/ndarray.h>
#include <nanobind/stl/array.h>
#include <nanobind/stl/optional.h>
#include <nanobind/stl/string.h>

namespace nb = nanobind;
using namespace nb::literals;

// Function to create a nanobind capsule for device memory
nb::capsule
create_device_capsule(void *ptr) {
    return nb::capsule(ptr, [](void *p) noexcept { hipFree(p); });
}

// Input types
using GridType = nb::ndarray<nb::pytorch, float, nb::ndim<3>, nb::device::cuda,
                             nb::c_contig>;
using CellType = nb::ndarray<nb::pytorch, float, nb::shape<-1, -1, -1, 3>,
                             nb::device::cuda, nb::c_contig>;
using AABBType = std::array<float, 6>;

// Output types
using VerticesType = nb::ndarray<nb::pytorch, float, nb::shape<-1, 3>,
                                 nb::device::cuda, nb::c_contig>;
using FacesType = nb::ndarray<nb::pytorch, int, nb::shape<-1, 3>,
                              nb::device::cuda, nb::c_contig>;

NB_MODULE(isoext_ext, m) {

    m.def(
        "marching_cubes",
        [](GridType grid, std::optional<AABBType> aabb,
           std::optional<CellType> cells, float level = 0.f, bool tight = true,
           std::string method = "nagae") {
            float *grid_ptr = grid.data();
            uint3 res = make_uint3(grid.shape(0), grid.shape(1), grid.shape(2));

            if (!aabb.has_value() && !cells.has_value()) {
                throw std::runtime_error(
                    "Either AABB or cell positions must be provided.");
            }
            if (aabb.has_value() && cells.has_value()) {
                throw std::runtime_error("Either AABB or cell positions must "
                                         "be provided, not both.");
            }

            thrust::device_vector<float3> cells_dv;
            float3 *cells_ptr = nullptr;

            if (aabb.has_value()) {
                auto a = aabb.value();
                cells_dv = get_cells_from_aabb(a, res);
                cells_ptr = thrust::raw_pointer_cast(cells_dv.data());
                tight = true;
            }

            if (cells.has_value()) {
                auto c = cells.value();
                cells_ptr = reinterpret_cast<float3 *>(c.data());
                for (int i = 0; i < 3; i++) {
                    if (grid.shape(i) != c.shape(i)) {
                        throw std::runtime_error(
                            "Resolutions of grid and cells must match except "
                            "for the last dimension of cells.");
                    }
                }
            }

            auto [v_ptr_raw, v_len, f_ptr_raw, f_len] = mc::marching_cubes(
                grid.data(), cells_ptr, res, level, tight, method);

            if (v_len == 0 || f_len == 0) {
                hipFree(v_ptr_raw);
                hipFree(f_ptr_raw);
                return nb::make_tuple(nb::none(), nb::none());
            }

            VerticesType v(v_ptr_raw, {v_len, 3},
                           create_device_capsule(v_ptr_raw));
            FacesType f(f_ptr_raw, {f_len, 3},
                        create_device_capsule(f_ptr_raw));

            return nb::make_tuple(v, f);
        },
        "grid"_a, "aabb"_a = nb::none(), "cells"_a = nb::none(),
        "level"_a = 0.f, "tight"_a = true, "method"_a = "nagae",
        "Marching Cubes");

    m.doc() = "A library for extracting iso-surfaces from level-set functions";
}
