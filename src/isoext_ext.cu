#include "hip/hip_runtime.h"
#include "mc.cuh"
#include "ndarray_types.cuh"
#include "utils.cuh"

#include <nanobind/nanobind.h>
#include <nanobind/ndarray.h>
#include <nanobind/stl/array.h>
#include <nanobind/stl/optional.h>
#include <nanobind/stl/string.h>

namespace nb = nanobind;
using namespace nb::literals;

// Function to create a nanobind capsule for device memory
nanobind::capsule
create_device_capsule(void *ptr) {
    return nanobind::capsule(ptr, [](void *p) noexcept { hipFree(p); });
}

NB_MODULE(isoext_ext, m) {

    m.def(
        "marching_cubes",
        [](GridType grid, std::optional<AABBType> aabb,
           std::optional<CellType> cells, float level = 0.f,
           std::string method = "nagae") {
            float *grid_ptr = grid.data();
            std::optional<float3 *> cells_ptr;
            if (cells.has_value()) {
                auto c = cells.value();
                cells_ptr = reinterpret_cast<float3 *>(c.data());
                for (int i = 0; i < 3; i++) {
                    if (grid.shape(i) != c.shape(i)) {
                        throw std::runtime_error(
                            "Resolutions of grid and cells must match except "
                            "for the last dimension of cells.");
                    }
                }
            }

            uint3 res = make_uint3(grid.shape(0), grid.shape(1), grid.shape(2));
            auto [v_ptr_raw, v_len, f_ptr_raw, f_len] = mc::marching_cubes(
                grid.data(), res, aabb, cells_ptr, level, method);

            if (v_len == 0 || f_len == 0) {
                hipFree(v_ptr_raw);
                hipFree(f_ptr_raw);
                return nb::make_tuple(nb::none(), nb::none());
            }

            VerticesType v(v_ptr_raw, {v_len, 3},
                           create_device_capsule(v_ptr_raw));
            FacesType f(f_ptr_raw, {f_len, 3},
                        create_device_capsule(f_ptr_raw));

            return nb::make_tuple(v, f);
        },
        "grid"_a, "aabb"_a = nb::none(), "cells"_a = nb::none(),
        "level"_a = 0.f, "method"_a = "nagae", "Marching Cubes");

    m.doc() = "A library for extracting iso-surfaces from level-set functions";
}
