#include "hip/hip_runtime.h"
#include "mc/nagae.cuh"
#include "shared_luts.cuh"

#include <thrust/device_vector.h>
#include <thrust/iterator/counting_iterator.h>

namespace mc {

namespace {
static MCRegistrar<Nagae> registrar("nagae");

struct process_cube_op {
    float3 *v;
    const uint8_t *cases;
    const uint *cell_indices;
    const float *values;
    const float3 *points;
    const uint *cells;
    const int *edges_table;
    const int *edge_status_table;
    const int *tri_table;
    const float level;

    process_cube_op(float3 *v, const uint8_t *cases, const uint *cell_indices,
                    const float *values, const float3 *points,
                    const uint *cells, const int *edges_table,
                    const int *edge_status_table, const int *tri_table,
                    const float level)
        : v(v), cases(cases), cell_indices(cell_indices), values(values),
          points(points), cells(cells), edges_table(edges_table),
          edge_status_table(edge_status_table), tri_table(tri_table),
          level(level) {}

    __host__ __device__ void operator()(uint idx) {
        uint32_t case_num = cases[idx];
        uint32_t cell_idx = cell_indices[idx];

        // Compute the location of each cube vertex.
        float3 c_p[8];
        float c_v[8];
        uint offset = cell_idx * 8;
        for (uint32_t i = 0; i < 8; i++) {
            c_p[i] = points[cells[offset + i]];
            c_v[i] = values[cells[offset + i]];
        }

        // Compute the intersection between the isosurface and each edge.
        int edge_status = edge_status_table[case_num];
        float3 cube_v[12];
        for (uint32_t i = 0; i < 12; i++) {
            if (edge_status & (1 << i)) {
                int p_0 = edges_table[i * 2];
                int p_1 = edges_table[i * 2 + 1];
                float denom = c_v[p_1] - c_v[p_0];
                float t = (denom != 0.0f) ? (level - c_v[p_0]) / denom : 0.0f;
                cube_v[i] = lerp(t, c_p[p_0], c_p[p_1]);
            }
        }

        // Assemble the triangles.
        case_num *= Nagae::max_len;   // max_length = 3 * 5 for Nagae
        uint32_t v0_idx = idx * Nagae::max_len;
        for (uint32_t i = 0; i < Nagae::max_len; i += 3) {
            uint32_t tri_idx = case_num + i;
            uint32_t v_idx = v0_idx + i;
            if (tri_table[tri_idx] != -1) {
                const float3 &v0 = cube_v[tri_table[tri_idx + 0]];
                const float3 &v1 = cube_v[tri_table[tri_idx + 1]];
                const float3 &v2 = cube_v[tri_table[tri_idx + 2]];

                if (v0 != v1 && v0 != v2 && v1 != v2) {
                    v[v_idx + 0] = v0;
                    v[v_idx + 1] = v1;
                    v[v_idx + 2] = v2;
                }
            } else {
                break;
            }
        }
    }
};
}   // anonymous namespace

void
Nagae::run(float3 *v, const uint num_cells, const uint8_t *cases,
           const uint *cell_indices, const float *values, const float3 *points,
           const uint *cells, const float level) {
    // Move the LUTs to the device.
    thrust::device_vector<int> edges_table_dv(edges_table,
                                              edges_table + edges_size);
    thrust::device_vector<int> edge_status_table_dv(
        edge_status_table, edge_status_table + edge_table_size);
    thrust::device_vector<int> tri_table_dv(
        Nagae::tri_table, Nagae::tri_table + Nagae::tri_table_size);

    thrust::for_each(thrust::counting_iterator<uint>(0),
                     thrust::counting_iterator<uint>(num_cells),
                     process_cube_op(v, cases, cell_indices, values, points,
                                     cells, edges_table_dv.data().get(),
                                     edge_status_table_dv.data().get(),
                                     tri_table_dv.data().get(), level));
}

}   // namespace mc