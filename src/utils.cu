#include "utils.cuh"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/copy.h>

template<typename T>
T* device_to_host(const T* d_ptr, size_t size) {
    T* h_ptr = (T*) malloc(size * sizeof(T));
    hipMemcpy(h_ptr, d_ptr, size * sizeof(T), hipMemcpyDeviceToHost);
    return h_ptr;
}

template<typename T>
T* host_to_device(const T* h_ptr, size_t size){
    T* d_ptr = (T*) malloc(size * sizeof(T));   
    hipMemcpy(d_ptr, h_ptr, size * sizeof(T), hipMemcpyHostToDevice);
    return d_ptr;
}

nanobind::capsule create_host_capsule(void* ptr) {
    return nanobind::capsule(ptr, [](void* p) noexcept {
        free(p);
    });
}

nanobind::capsule create_device_capsule(void* ptr) {
    return nanobind::capsule(ptr, [](void* p) noexcept {
        hipFree(p);
    });
}