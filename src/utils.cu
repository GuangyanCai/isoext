#include "hip/hip_runtime.h"
#include "math.cuh"
#include "utils.cuh"

#include <thrust/binary_search.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/unique.h>

void
vertex_welding(thrust::device_vector<float3> &v, thrust::device_vector<int> &f,
               bool skip_scatter) {

    thrust::device_vector<float3> sorted_v;

    if (skip_scatter) {
        sorted_v = v;
    } else {
        // Scatter v to sorted_v based on f
        thrust::scatter(v.begin(), v.end(), f.begin(), sorted_v.begin());
        f.clear();
        f.resize(v.size());
        thrust::sequence(f.begin(), f.end());
    }

    // Remove duplicated vertices
    thrust::sort(sorted_v.begin(), sorted_v.end(), float3_less_pred());
    sorted_v.erase(
        thrust::unique(sorted_v.begin(), sorted_v.end(), float3_elem_eq_pred()),
        sorted_v.end());

    thrust::lower_bound(sorted_v.begin(), sorted_v.end(), v.begin(), v.end(),
                        f.begin(), float3_less_pred());

    // Update vertex array
    v = std::move(sorted_v);
}
